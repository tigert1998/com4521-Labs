#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""

#define N 65536
#define THREADS_PER_BLOCK 128

void checkCUDAError(const char *);
void random_ints(int *a);

__device__ int d_a[N], d_b[N], d_c[N];

__global__ void vectorAdd(int max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < max) d_c[i] = d_a[i] + d_b[i];
}

void QueryDevices() {
  int n;

  hipGetDeviceCount(&n);
  for (int i = 0; i < n; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) * 1e3 /
               (1 << 30));
  }
}

int main(void) {
  QueryDevices();

  int *a, *b, *c, *c_ref;  // host copies of a, b, c
  unsigned int size = N * sizeof(int);

  // Alloc space for host copies of a, b, c and setup input values
  a = (int *)malloc(size);
  random_ints(a);
  b = (int *)malloc(size);
  random_ints(b);
  c = (int *)malloc(size);
  c_ref = (int *)malloc(size);
  for (int i = 0; i < N; i++) c_ref[i] = a[i] + b[i];

  hipMemcpyToSymbol(HIP_SYMBOL(d_a), a, size);
  hipMemcpyToSymbol(HIP_SYMBOL(d_b), b, size);
  checkCUDAError("CUDA memcpy");

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  // Launch add() kernel on GPU
  vectorAdd<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,
              THREADS_PER_BLOCK>>>(N);
  hipEventRecord(stop);
  checkCUDAError("CUDA kernel");

  hipEventSynchronize(stop);
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  printf("vectorAdd takes %.3lfms\n", ms);
  printf("Measured Global Memory Bandwidth (GB/s): %f\n",
         size * 3 / ms * 1e3 / (1 << 30));

  // Copy result back to host
  hipMemcpyFromSymbol(c, HIP_SYMBOL(d_c), size);
  checkCUDAError("CUDA memcpy");
  for (int i = 0; i < N; i++)
    if (c_ref[i] != c[i]) {
      puts("FAIL");
      break;
    }

  // Cleanup
  free(a);
  free(b);
  free(c);

  return 0;
}

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

void random_ints(int *a) {
  for (unsigned int i = 0; i < N; i++) {
    a[i] = rand();
  }
}
