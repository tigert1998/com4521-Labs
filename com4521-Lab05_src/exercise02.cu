#include "hip/hip_runtime.h"
/*
 * Source code for this lab class is modifed from the book CUDA by Exmaple and
 * provided by permission of NVIDIA Corporation
 */

#include <hip/hip_runtime.h>
#include <>
#include <vector_functions.h>
#include <hip/hip_vector_types.h>

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <string>

#include "common.cuh"

#define IMAGE_DIM 2048
#define MAX_SPHERES 2048

#define rnd(x) (x * rand() / RAND_MAX)
#define INF 2e10f

void OutputImageFile(uchar4 *image, const std::string &filename);
void CheckCUDAError(const char *msg);

struct Sphere {
  float r, b, g;
  float radius;
  float x, y, z;
};

/* Device Code */

__constant__ unsigned int d_sphere_count;

__constant__ Sphere const_spheres[MAX_SPHERES];

#define SPHERE_INTERSECT(s, ox, oy, t, n)                        \
  do {                                                           \
    float dx = ox - s.x;                                         \
    float dy = oy - s.y;                                         \
    if (dx * dx + dy * dy < s.radius * s.radius) {               \
      float dz = sqrtf(s.radius * s.radius - dx * dx - dy * dy); \
      n = dz / s.radius;                                         \
      t = dz + s.z;                                              \
    } else {                                                     \
      t = -INF;                                                  \
    }                                                            \
  } while (0)

#define RAY_TRACE(image, spheres)                  \
  do {                                             \
    int x = threadIdx.x + blockIdx.x * blockDim.x; \
    int y = threadIdx.y + blockIdx.y * blockDim.y; \
    int offset = x + y * blockDim.x * gridDim.x;   \
    float ox = (x - IMAGE_DIM / 2.0f);             \
    float oy = (y - IMAGE_DIM / 2.0f);             \
    float r = 0, g = 0, b = 0;                     \
    float maxz = -INF;                             \
    for (int i = 0; i < d_sphere_count; i++) {     \
      float n, t;                                  \
      SPHERE_INTERSECT(spheres[i], ox, oy, t, n);  \
      if (t > maxz) {                              \
        float fscale = n;                          \
        r = spheres[i].r * fscale;                 \
        g = spheres[i].g * fscale;                 \
        b = spheres[i].b * fscale;                 \
        maxz = t;                                  \
      }                                            \
    }                                              \
    image[offset].x = (int)(r * 255);              \
    image[offset].y = (int)(g * 255);              \
    image[offset].z = (int)(b * 255);              \
    image[offset].w = 255;                         \
  } while (0)

__global__ void RayTraceReadOnly(uchar4 *image,
                                 const Sphere *__restrict__ spheres) {
  RAY_TRACE(image, spheres);
}

__global__ void RayTraceConst(uchar4 *image) {
  RAY_TRACE(image, const_spheres);
}

__global__ void RayTraceNormal(uchar4 *image, Sphere *spheres) {
  RAY_TRACE(image, spheres);
}

/* Host code */

int main(void) {
  unsigned int image_size = IMAGE_DIM * IMAGE_DIM * sizeof(uchar4);
  unsigned int spheres_size = MAX_SPHERES * sizeof(Sphere);

  float3 timing_data;  // timing data where [0]=normal, [1]=read-only, [2]=const
  uchar4 *h_image, *d_image;
  Sphere *h_s, *d_s;

  h_s = (Sphere *)malloc(spheres_size);

  // allocate memory on the GPU for the output image
  hipMalloc((void **)&d_image, image_size);
  CheckCUDAError("CUDA malloc");
  hipMalloc(&d_s, spheres_size);
  CheckCUDAError("CUDA malloc");

  // create some random spheres
  for (int i = 0; i < MAX_SPHERES; i++) {
    h_s[i].r = rnd(1.0f);
    h_s[i].g = rnd(1.0f);
    h_s[i].b = rnd(1.0f);
    h_s[i].x = rnd((float)IMAGE_DIM) - (IMAGE_DIM / 2.0f);
    h_s[i].y = rnd((float)IMAGE_DIM) - (IMAGE_DIM / 2.0f);
    h_s[i].z = rnd((float)IMAGE_DIM) - (IMAGE_DIM / 2.0f);
    h_s[i].radius = rnd(100.0f) + 20;
  }

  // copy to device memory
  hipMemcpy(d_s, h_s, spheres_size, hipMemcpyHostToDevice);
  CheckCUDAError("CUDA memcpy");

  // generate host image
  h_image = (uchar4 *)malloc(image_size);

  // cuda layout
  dim3 blocksPerGrid(IMAGE_DIM / 16, IMAGE_DIM / 16);
  dim3 threadsPerBlock(16, 16);

  // output timings
  printf("Timing Data Table\n Spheres | Normal | Read-only | Const\n");
  for (unsigned int sphere_count = 16; sphere_count <= MAX_SPHERES;
       sphere_count *= 2) {
    hipMemcpyToSymbol(HIP_SYMBOL(d_sphere_count), &sphere_count, sizeof(unsigned int));
    CheckCUDAError("CUDA copy sphere count to device");

    // generate a image from the sphere data

    TIME("kernel (normal)", timing_data.x, RayTraceNormal, blocksPerGrid,
         threadsPerBlock, d_image, d_s);

    hipMemcpy(h_image, d_image, image_size, hipMemcpyDeviceToHost);
    CheckCUDAError("CUDA memcpy from device");
    OutputImageFile(h_image, std::string("normal.") +
                                 std::to_string(sphere_count) + ".ppm");

    TIME("kernel (read only)", timing_data.y, RayTraceReadOnly, blocksPerGrid,
         threadsPerBlock, d_image, d_s);

    hipMemcpy(h_image, d_image, image_size, hipMemcpyDeviceToHost);
    CheckCUDAError("CUDA memcpy from device");
    OutputImageFile(h_image, std::string("readonly.") +
                                 std::to_string(sphere_count) + ".ppm");

    TIME("kernel (constant)", timing_data.z, RayTraceConst, blocksPerGrid,
         threadsPerBlock, d_image);

    hipMemcpy(h_image, d_image, image_size, hipMemcpyDeviceToHost);
    CheckCUDAError("CUDA memcpy from device");
    OutputImageFile(h_image, std::string("constant.") +
                                 std::to_string(sphere_count) + ".ppm");

    printf(" %-7i | %-6.3f | %-9.3f | %.3f\n", sphere_count, timing_data.x,
           timing_data.y, timing_data.z);
  }

  hipFree(d_image);
  free(h_image);
  free(h_s);

  return 0;
}

void OutputImageFile(uchar4 *image, const std::string &filename) {
  FILE *f;  // output file handle

  // open the output file and write header info for PPM filetype
  f = fopen(filename.c_str(), "wb");
  if (f == NULL) {
    fprintf(stderr, "Error opening 'output.ppm' output file\n");
    exit(1);
  }
  fprintf(f, "P6\n");
  fprintf(f, "# COM4521 Lab 05 Exercise02\n");
  fprintf(f, "%d %d\n%d\n", IMAGE_DIM, IMAGE_DIM, 255);
  for (int x = 0; x < IMAGE_DIM; x++) {
    for (int y = 0; y < IMAGE_DIM; y++) {
      int i = x + y * IMAGE_DIM;
      fwrite(&image[i], sizeof(unsigned char), 3,
             f);  // only write rgb (ignoring a)
    }
  }

  fclose(f);
}

void CheckCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
