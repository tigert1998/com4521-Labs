#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"


#define N 2050
#define THREADS_PER_BLOCK 128

void checkCUDAError(const char *);
void random_ints(int *a);

__global__ void vectorAdd(int *a, int *b, int *c, int max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= max) return;
  c[i] = a[i] + b[i];
}

void vectorAddCPU(int *a, int *b, int *c, int max) {
  for (int i = 0; i < max; i++) c[i] = a[i] + b[i];
}

bool validate(int *c, int *c_ref, int max) {
  for (int i = 0; i < max; i++)
    if (c[i] != c_ref[i]) {
      return false;
    }
  return true;
}

int main(void) {
  int *a, *b, *c, *c_ref;  // host copies of a, b, c
  int *d_a, *d_b, *d_c;    // device copies of a, b, c
  int errors;
  unsigned int size = N * sizeof(int);

  // Alloc space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  checkCUDAError("CUDA malloc");

  // Alloc space for host copies of a, b, c and setup input values
  a = (int *)malloc(size);
  random_ints(a);
  b = (int *)malloc(size);
  random_ints(b);
  c = (int *)malloc(size);
  c_ref = (int *)malloc(size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  checkCUDAError("CUDA memcpy");

  // Launch add() kernel on GPU
  vectorAdd<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,
              THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);
  checkCUDAError("CUDA kernel");

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  checkCUDAError("CUDA memcpy");

  vectorAddCPU(a, b, c_ref, N);
  if (validate(c, c_ref, N)) {
    puts("PASS");
  } else {
    puts("FAIL");
  }

  // Cleanup
  free(a);
  free(b);
  free(c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  checkCUDAError("CUDA cleanup");

  return 0;
}

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

void random_ints(int *a) {
  for (unsigned int i = 0; i < N; i++) {
    a[i] = rand();
  }
}
